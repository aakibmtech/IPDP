#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <random>
#include <iostream>
#include <chrono>
#include <algorithm>
#include <numeric>
#include <functional>
#include <mutex>
#include <atomic>

//==============================================================================
// Event System
//==============================================================================

enum class PreprocessingEvent {
    NOISE_DETECTED,
    CLEANING_COMPLETED,
    PROGRESS_UPDATE
};

struct EventData {
    int recordIndex;
    int parameterIndex;
    int recordsProcessed;
};

class EventHandler {
private:
    std::function<void(PreprocessingEvent, const EventData&)> eventCallback;
    std::mutex eventMutex;

public:
    void setEventCallback(std::function<void(PreprocessingEvent, const EventData&)> callback) {
        std::lock_guard<std::mutex> lock(eventMutex);
        eventCallback = callback;
    }

    void triggerEvent(PreprocessingEvent event, const EventData& data) {
        std::lock_guard<std::mutex> lock(eventMutex);
        if (eventCallback) {
            eventCallback(event, data);
        }
    }
};

//==============================================================================
// Data Structures
//==============================================================================

struct PreprocessingProgress {
    std::atomic<int> processedRecords{0};
    int totalRecords;
    std::atomic<int> cleanedRecords{0};
};

// Global progress tracking
static std::atomic<int> globalProcessedRecords{0};
static std::atomic<int> globalCleanedRecords{0};
static EventHandler globalEventHandler;

//==============================================================================
// CUDA Kernel Functions
//==============================================================================

__global__ void calculateMediansKernel(double* data, double* medians, int chunkSize, int numParameters) {
    int paramIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (paramIdx >= numParameters) return;

    // Collect valid values for each parameter
    extern __shared__ double sharedValues[];
    int validCount = 0;
    for (int i = 0; i < chunkSize; ++i) {
        double value = data[i * numParameters + paramIdx];
        if (value != -1) {
            sharedValues[validCount++] = value;
        }
    }

    // Sort the valid values
    for (int i = 0; i < validCount - 1; ++i) {
        for (int j = i + 1; j < validCount; ++j) {
            if (sharedValues[i] > sharedValues[j]) {
                double temp = sharedValues[i];
                sharedValues[i] = sharedValues[j];
                sharedValues[j] = temp;
            }
        }
    }

    // Calculate median
    if (validCount == 0) {
        medians[paramIdx] = 0.0;
    } else if (validCount % 2 == 0) {
        medians[paramIdx] = (sharedValues[validCount / 2 - 1] + sharedValues[validCount / 2]) / 2.0;
    } else {
        medians[paramIdx] = sharedValues[validCount / 2];
    }
}

__global__ void processRecordsKernel(double* data, double* medians, int chunkSize, int numParameters, int* noiseCount) {
    int recordIdx = blockIdx.x;
    int paramIdx = threadIdx.x;

    if (recordIdx >= chunkSize || paramIdx >= numParameters) return;

    int baseIdx = recordIdx * numParameters;
    int idx = baseIdx + paramIdx;
    if (data[idx] == -1) {
        data[idx] = medians[paramIdx];
        atomicAdd(noiseCount, 1);
    }
}

//==============================================================================
// Optimized Data Preprocessor Class
//==============================================================================

class DataPreprocessor {
public:
    static const int NUM_PARAMETERS = 50;
    static const int CHUNK_SIZE = 1000;
    static const int TOTAL_RECORDS = 1000000;

private:
    std::vector<double> data;
    std::vector<double> medians;
    int rank;
    int size;
    PreprocessingProgress progress;

    int noiseCount;

    double* d_data;
    double* d_medians;
    int* d_noiseCount;

    void calculateMedians() {
        // Allocate device memory for medians and shared memory size
        hipMalloc(&d_medians, NUM_PARAMETERS * sizeof(double));
        size_t sharedMemorySize = CHUNK_SIZE * sizeof(double);

        // Launch kernel to calculate medians
        int threadsPerBlock = 256;
        //int blocksPerGrid = (NUM_PARAMETERS + threadsPerBlock - 1) / threadsPerBlock;
        int blocksPerGrid = 65535;
	calculateMediansKernel<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(d_data, d_medians, CHUNK_SIZE, NUM_PARAMETERS);

        // Copy medians back to host
        hipMemcpy(medians.data(), d_medians, NUM_PARAMETERS * sizeof(double), hipMemcpyDeviceToHost);
    }

    void processRecords() {
        // Allocate memory for noise count on device
        hipMalloc(&d_noiseCount, sizeof(int));
        hipMemset(d_noiseCount, 0, sizeof(int));

        // Launch kernel to process records
        dim3 threadsPerBlock(NUM_PARAMETERS);
        dim3 numBlocks(CHUNK_SIZE);
        processRecordsKernel<<<numBlocks, threadsPerBlock>>>(d_data, d_medians, CHUNK_SIZE, NUM_PARAMETERS, d_noiseCount);

        // Copy noise count back to host
        hipMemcpy(&noiseCount, d_noiseCount, sizeof(int), hipMemcpyDeviceToHost);

        globalCleanedRecords += noiseCount;
    }

public:
    DataPreprocessor(int _rank, int _size) 
        : rank(_rank), size(_size), noiseCount(0) {
        progress.totalRecords = TOTAL_RECORDS;
        data.resize(CHUNK_SIZE * NUM_PARAMETERS);
        medians.resize(NUM_PARAMETERS);

        // Allocate device memory for data
        hipMalloc(&d_data, CHUNK_SIZE * NUM_PARAMETERS * sizeof(double));
    }

    ~DataPreprocessor() {
        hipFree(d_data);
        hipFree(d_medians);
        hipFree(d_noiseCount);
    }

    void processChunkParallel() {
        // Copy data to device
        hipMemcpy(d_data, data.data(), CHUNK_SIZE * NUM_PARAMETERS * sizeof(double), hipMemcpyHostToDevice);

        // Calculate medians first
        calculateMedians();

        // Process records in parallel
        processRecords();

        // Update global progress
        int records_processed = CHUNK_SIZE;
        globalProcessedRecords += records_processed;

        // Gather total progress across all processes
        int local_processed = globalProcessedRecords.load();
        int total_processed;
        MPI_Allreduce(&local_processed, &total_processed, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);

        EventData progressData = {0, 0, total_processed};
        globalEventHandler.triggerEvent(PreprocessingEvent::PROGRESS_UPDATE, progressData);
    }

    std::vector<double>& getData() { return data; }
    void setData(const std::vector<double>& newData) { data = newData; }
};

//==============================================================================
// Main Function with Optimized Implementation
//==============================================================================

int main(int argc, char** argv) {
    int provided;
    MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
    if (provided != MPI_THREAD_MULTIPLE) {
        std::cerr << "Thread multiple support unavailable\n";
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    // Set up event handling
    globalEventHandler.setEventCallback(
        [rank](PreprocessingEvent event, const EventData& data) {
            if (rank == 0 && event == PreprocessingEvent::PROGRESS_UPDATE) {
                std::cout << "Total records processed: " << data.recordsProcessed << std::endl;
            }
        }
    );

    const int total_chunks = DataPreprocessor::TOTAL_RECORDS / DataPreprocessor::CHUNK_SIZE;
    const int chunks_per_process = total_chunks / size;


    // Only root generates full dataset
    std::vector<double> fullData;
    if (rank == 0) {
        fullData.resize(DataPreprocessor::TOTAL_RECORDS * DataPreprocessor::NUM_PARAMETERS);
        
        #pragma omp parallel
        {
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_int_distribution<> dis(0, 99);
            std::uniform_real_distribution<> noise_dis(0, 1);
            
            #pragma omp for schedule(dynamic)
            for (int i = 0; i < DataPreprocessor::TOTAL_RECORDS; i++) {
                for (int j = 0; j < DataPreprocessor::NUM_PARAMETERS; j++) {
                    int idx = i * DataPreprocessor::NUM_PARAMETERS + j;
                    fullData[idx] = (noise_dis(gen) < 0.05) ? -1 : dis(gen);
                }
            }
        }
    }

    // Process chunks
    DataPreprocessor preprocessor(rank, size);
    std::vector<double> chunk(DataPreprocessor::CHUNK_SIZE * DataPreprocessor::NUM_PARAMETERS);
    auto startTime = std::chrono::high_resolution_clock::now();
    for (int chunk_idx = 0; chunk_idx < chunks_per_process; chunk_idx++) {
        if (rank == 0) {
            // Distribute chunks
            for (int p = 0; p < size; p++) {
                int base_chunk_idx = chunk_idx * size + p;
                int start_idx = base_chunk_idx * DataPreprocessor::CHUNK_SIZE * 
                              DataPreprocessor::NUM_PARAMETERS;

                if (p == 0) {
                    std::copy(fullData.begin() + start_idx,
                             fullData.begin() + start_idx + chunk.size(),
                             chunk.begin());
                } else {
                    MPI_Send(&fullData[start_idx], chunk.size(), MPI_DOUBLE, 
                            p, 0, MPI_COMM_WORLD);
                }
            }
        } else {
            MPI_Recv(chunk.data(), chunk.size(), MPI_DOUBLE, 
                    0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }

        preprocessor.setData(chunk);
        preprocessor.processChunkParallel();
    }

    // Gather statistics
    int local_cleaned = globalCleanedRecords.load();
    int total_cleaned;
    MPI_Reduce(&local_cleaned, &total_cleaned, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);

    auto endTime = std::chrono::high_resolution_clock::now();
    double processingTime = std::chrono::duration<double>(endTime - startTime).count();

    if (rank == 0) {
        std::vector<double> allTimes(size);
        MPI_Gather(&processingTime, 1, MPI_DOUBLE, allTimes.data(), 1, MPI_DOUBLE, 
                  0, MPI_COMM_WORLD);
        
        double totalTime = *std::max_element(allTimes.begin(), allTimes.end());
        double avgTime = std::accumulate(allTimes.begin(), allTimes.end(), 0.0) / size;

        std::cout << "\nFinal Performance Metrics:\n"
                  << "Total Processing Time: " << totalTime << " seconds\n"
                  << "Average Time per Process: " << avgTime << " seconds\n"
                  << "Total chunks processed: " << total_chunks << "\n"
                  << "Chunks per process: " << chunks_per_process << "\n"
                  << "Total noisy records cleaned: " << total_cleaned << "\n";
    } else {
        MPI_Gather(&processingTime, 1, MPI_DOUBLE, nullptr, 0, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    }

    MPI_Finalize();
    return 0;
}

